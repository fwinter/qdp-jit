#include "hip/hip_runtime.h"
#include<vector>
#include<iostream>
#include<utility>
#include<algorithm>
#include<map>

#include <cuda_special.h>

namespace {

  __device__ int getGlobalIdx_2D_1D()
  {
    int blockId = blockIdx.y* gridDim.x+ blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
  }




  class hipComplex
  {
  public:
  
    __device__ hipComplex() {}
  
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}

    __device__ float real() const { return r; }

    __device__ float imag() const { return i; }

    __device__ hipComplex& operator+=(const hipComplex& a) {
      r = r + a.real();
      i = i + a.imag();
      return *this;
    }

    __device__ hipComplex& operator=(const hipComplex& a) {
      r = a.real();
      i = a.imag();
      return *this;
    }

  
  private:
    float   r;
    float   i;
  };

  __device__ hipComplex adj( const hipComplex& a )
  {
    return hipComplex( a.real() , -a.imag() );
  }

  
  __device__ hipComplex operator*( const hipComplex& a , const hipComplex& b )
  {
    return hipComplex(a.real()*b.real() - a.imag()*b.imag(), a.imag()*b.real() + a.real()*b.imag());
  }

#if 0
  __device__ hipComplex operator+( const hipComplex& a , const hipComplex& b )
  {
    return hipComplex( a.real() + b.real() , a.imag() + b.imag() );
  }

  __device__ hipComplex operator-( const hipComplex& a , const hipComplex& b )
  {
    return hipComplex( a.real() - b.real() , a.imag() - b.imag() );
  }
#endif

  //
  // QDP-JIT datalayout
  //
  __device__ int iu( int N , int c1 , int c2 , int z, int idx)
  {
    return ((z * 3 + c1) * 3 + c2) * N + idx;
  }

  __device__ int iQ( int N , int s1 , int s2 , int c1 , int c2 , int z, int idx)
  {
    return ((((z * 3 + c1) * 3 + c2) * 4 + s1 ) * 4 + s2 ) * N + idx;
  }

  __device__ int iQscalar( int s1 , int s2 , int c1 , int c2 , int z, int idx)
  {
    return ((((idx * 4 + s1) * 4 + s2) * 3 + c1) * 3 + c2 ) * 2 + z;
  }



} // namespace




__global__ void cuda_prop_deriv( int threads, int N , int start ,
				 float* dest,
				 float* Q1,
				 float* Q2,
				 float* u1,
				 float* u2,
				 int* goff0m, float* recv0m,  
				 int* goff0p, float* recv0p,
				 bool do_site_perm, int* site_perm
				 )
{
  int th_idx = getGlobalIdx_2D_1D();
  if (th_idx >= threads)
    return;

  int idx = th_idx + start;

  if (do_site_perm)
    idx = site_perm[ idx ];


  hipComplex Qs[3][3];
  hipComplex us[3][3];
  hipComplex res[3][3];

  //hipComplex psil[3][3];

  for ( int spin_i = 0 ; spin_i < 4 ; ++spin_i )
    {
      for ( int spin_j = 0 ; spin_j < 4 ; ++spin_j )
  	{
	  // First term
	  // u[mu] * shift( Q1 , FORWARD , mu )
	  
	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  if (goff0m[idx] < 0)
		    Qs[c1][c2] = hipComplex(  recv0m[ iQscalar(spin_i,spin_j,c1,c2,0,-goff0m[idx]-1) ] , recv0m[ iQscalar(spin_i,spin_j,c1,c2,1,-goff0m[idx]-1) ] );
		  else
		    Qs[c1][c2] = hipComplex(  Q1[ iQ(N,spin_i,spin_j,c1,c2,0,goff0m[idx]) ] , Q1[ iQ(N,spin_i,spin_j,c1,c2,1,goff0m[idx]) ] );

		  us[c1][c2] = hipComplex(  u1[ iu(N,c1,c2,0,idx) ] , u1[ iu(N,c1,c2,1,idx) ] );
		}
	    }

	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  res[c1][c2] = us[c1][0] * Qs[0][c2];
	
		  for( int k = 1 ; k < 3 ; ++k )
		    res[c1][c2] += us[c1][k] * Qs[k][c2];
		}
	    }
	      
  
	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  dest[ iQ(N,spin_i,spin_j,c1,c2,0,idx) ] -= res[c1][c2].real();
		  dest[ iQ(N,spin_i,spin_j,c1,c2,1,idx) ] -= res[c1][c2].imag();
		}
	    }

	  
	  // Second term
	  // shift( adj( u[mu] ) * Q2 , BACKWARD , mu )
	  
	  if (goff0p[idx] < 0)
	    {
	      for( int c1 = 0 ; c1 < 3 ; ++c1 )
		{
		  for( int c2 = 0 ; c2 < 3 ; ++c2)
		    {
		      res[c1][c2] = hipComplex(  recv0p[ iQscalar(spin_i,spin_j,c1,c2,0,-goff0p[idx]-1) ] , recv0p[ iQscalar(spin_i,spin_j,c1,c2,1,-goff0p[idx]-1) ] );
		    }
		}
	    }
	  else
	    {
	      for( int c1 = 0 ; c1 < 3 ; ++c1 )
		{
		  for( int c2 = 0 ; c2 < 3 ; ++c2)
		    {
		      Qs[c1][c2] = hipComplex(  Q2[ iQ(N,spin_i,spin_j,c1,c2,0,goff0p[idx]) ] , Q2[ iQ(N,spin_i,spin_j,c1,c2,1,goff0p[idx]) ] );

		      us[c1][c2] = hipComplex(  u2[ iu(N,c1,c2,0,goff0p[idx]) ] , u2[ iu(N,c1,c2,1,goff0p[idx]) ] );
		    }
		}

	      for( int c1 = 0 ; c1 < 3 ; ++c1 )
		{
		  for( int c2 = 0 ; c2 < 3 ; ++c2)
		    {
		      res[c1][c2] = adj( us[0][c1] ) * Qs[0][c2];
	
		      for( int k = 1 ; k < 3 ; ++k )
			res[c1][c2] += adj( us[k][c1] ) * Qs[k][c2];
		    }
		}
	    }

	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  dest[ iQ(N,spin_i,spin_j,c1,c2,0,idx) ] -= res[c1][c2].real();
		  dest[ iQ(N,spin_i,spin_j,c1,c2,1,idx) ] -= res[c1][c2].imag();
		}
	    }

	  
	} // spin_j
    } // spin_i
}






void evaluate_special_prop_deriv( int threads, int N , int start, std::vector<void*> args , bool do_siteperm , int func_num )
{
  const int default_blocksize = 128;

  jumper_jit_stats_special(func_num);
  
  if (cuda_special_get_maxgridx() == -1)
    {
      std::cerr << "evaluate_dslash, cuda_special_maxgridx not set\n";
      exit(1);
    }
  
  int threads_per_block = default_blocksize;
  if ( cuda_special_get_blocksize().count( func_num ) > 0 )
    threads_per_block = cuda_special_get_blocksize()[ func_num ];
  
  std::pair<int,int> size = getBlockDim( N , threads_per_block );
  dim3 grid(  size.first , size.second , 1 );
  dim3 block( threads_per_block , 1 , 1 );

  // std::cout << "launching : grid( " << size.first << " , " << size.second << " , 1)   ";
  // std::cout << "launching : block( " << threads_per_block << " , 1 , 1 )\n";

  // 0 dest 
  // 1 u
  // 2,3 shift (off,recv)
  // 4 Q

  // 5,6 shift
  // 7 u
  // 8 Q

  float* dest = (float*)*(void**)args[0];

  int* goff[2];
  goff[0] = (int*)*(void**)args[2];
  goff[1] = (int*)*(void**)args[5];

  float* recv[2];
  recv[0] = (float*)*(void**)args[3];
  recv[1] = (float*)*(void**)args[6];

  float* Q1 = (float*)*(void**)args[4];
  float* Q2 = (float*)*(void**)args[8];

  float* u1 = (float*)*(void**)args[1];
  float* u2 = (float*)*(void**)args[7];

  int* site_perm = do_siteperm ? (int*)*(void**)args[9] : NULL;

#if 1
  cuda_prop_deriv<<< grid , block >>>( threads, N , start ,
				       dest ,
				       Q1 , Q2,
				       u1 , u1,
				       goff[0] , recv[0] , 
				       goff[1] , recv[1] , 
				       do_siteperm, site_perm
				       );
#endif

}

