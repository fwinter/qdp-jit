#include "hip/hip_runtime.h"
#include<vector>
#include<iostream>
#include<utility>
#include<algorithm>
#include<map>

#include <cuda_special.h>

namespace {

  __device__ int getGlobalIdx_2D_1D()
  {
    int blockId = blockIdx.y* gridDim.x+ blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
  }



  template<class T>
  class hipComplex
  {
  public:
  
    __device__ hipComplex() {}
  
    __device__ hipComplex( T a, T b ) : r(a), i(b)  {}

    __device__ T real() const { return r; }

    __device__ T imag() const { return i; }

    __device__ hipComplex& operator+=(const hipComplex& a) {
      r = r + a.real();
      i = i + a.imag();
      return *this;
    }

    __device__ hipComplex& operator=(const hipComplex& a) {
      r = a.real();
      i = a.imag();
      return *this;
    }

  
  private:
    T   r;
    T   i;
  };

  template<class T>
  __device__ hipComplex<T> adj( const hipComplex<T>& a )
  {
    return hipComplex<T>( a.real() , -a.imag() );
  }


  template<class T>
  __device__ hipComplex<T> operator*( const hipComplex<T>& a , const hipComplex<T>& b )
  {
    return hipComplex<T>(a.real()*b.real() - a.imag()*b.imag(), a.imag()*b.real() + a.real()*b.imag());
  }

#if 0
  __device__ hipComplex operator+( const hipComplex& a , const hipComplex& b )
  {
    return hipComplex( a.real() + b.real() , a.imag() + b.imag() );
  }

  __device__ hipComplex operator-( const hipComplex& a , const hipComplex& b )
  {
    return hipComplex( a.real() - b.real() , a.imag() - b.imag() );
  }
#endif

  //
  // QDP-JIT datalayout
  //
  __device__ int iu( int N , int c1 , int c2 , int z, int idx)
  {
    return ((z * 3 + c1) * 3 + c2) * N + idx;
  }

  __device__ int iQ( int N , int s1 , int s2 , int c1 , int c2 , int z, int idx)
  {
    return ((((z * 3 + c1) * 3 + c2) * 4 + s1 ) * 4 + s2 ) * N + idx;
  }

  __device__ int iQscalar( int s1 , int s2 , int c1 , int c2 , int z, int idx)
  {
    return ((((idx * 4 + s1) * 4 + s2) * 3 + c1) * 3 + c2 ) * 2 + z;
  }



} // namespace



template<class T>
__global__ void cuda_prop_deriv( int threads, int N , int start ,
				 T* dest,
				 T* Q1,
				 T* Q2,
				 T* u1,
				 T* u2,
				 int* goff0m, T* recv0m,  
				 int* goff0p, T* recv0p,
				 bool do_site_perm, int* site_perm
				 )
{
  int th_idx = getGlobalIdx_2D_1D();
  if (th_idx >= threads)
    return;

  int idx = th_idx + start;

  if (do_site_perm)
    idx = site_perm[ idx ];


  hipComplex<T> Qs[3][3];
  hipComplex<T> us[3][3];
  hipComplex<T> res[3][3];


  for ( int spin_i = 0 ; spin_i < 4 ; ++spin_i )
    {
      for ( int spin_j = 0 ; spin_j < 4 ; ++spin_j )
  	{
	  // First term
	  // u[mu] * shift( Q1 , FORWARD , mu )
	  
	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  if (goff0m[idx] < 0)
		    Qs[c1][c2] = hipComplex<T>(  recv0m[ iQscalar(spin_i,spin_j,c1,c2,0,-goff0m[idx]-1) ] , recv0m[ iQscalar(spin_i,spin_j,c1,c2,1,-goff0m[idx]-1) ] );
		  else
		    Qs[c1][c2] = hipComplex<T>(  Q1[ iQ(N,spin_i,spin_j,c1,c2,0,goff0m[idx]) ] , Q1[ iQ(N,spin_i,spin_j,c1,c2,1,goff0m[idx]) ] );

		  us[c1][c2] = hipComplex<T>(  u1[ iu(N,c1,c2,0,idx) ] , u1[ iu(N,c1,c2,1,idx) ] );
		}
	    }

	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  res[c1][c2] = us[c1][0] * Qs[0][c2];
	
		  for( int k = 1 ; k < 3 ; ++k )
		    res[c1][c2] += us[c1][k] * Qs[k][c2];
		}
	    }
	      
  
	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  dest[ iQ(N,spin_i,spin_j,c1,c2,0,idx) ] -= res[c1][c2].real();
		  dest[ iQ(N,spin_i,spin_j,c1,c2,1,idx) ] -= res[c1][c2].imag();
		}
	    }

	  
	  // Second term
	  // shift( adj( u[mu] ) * Q2 , BACKWARD , mu )
	  
	  if (goff0p[idx] < 0)
	    {
	      for( int c1 = 0 ; c1 < 3 ; ++c1 )
		{
		  for( int c2 = 0 ; c2 < 3 ; ++c2)
		    {
		      res[c1][c2] = hipComplex<T>(  recv0p[ iQscalar(spin_i,spin_j,c1,c2,0,-goff0p[idx]-1) ] , recv0p[ iQscalar(spin_i,spin_j,c1,c2,1,-goff0p[idx]-1) ] );
		    }
		}
	    }
	  else
	    {
	      for( int c1 = 0 ; c1 < 3 ; ++c1 )
		{
		  for( int c2 = 0 ; c2 < 3 ; ++c2)
		    {
		      Qs[c1][c2] = hipComplex<T>(  Q2[ iQ(N,spin_i,spin_j,c1,c2,0,goff0p[idx]) ] , Q2[ iQ(N,spin_i,spin_j,c1,c2,1,goff0p[idx]) ] );

		      us[c1][c2] = hipComplex<T>(  u2[ iu(N,c1,c2,0,goff0p[idx]) ] , u2[ iu(N,c1,c2,1,goff0p[idx]) ] );
		    }
		}

	      for( int c1 = 0 ; c1 < 3 ; ++c1 )
		{
		  for( int c2 = 0 ; c2 < 3 ; ++c2)
		    {
		      res[c1][c2] = adj( us[0][c1] ) * Qs[0][c2];
	
		      for( int k = 1 ; k < 3 ; ++k )
			res[c1][c2] += adj( us[k][c1] ) * Qs[k][c2];
		    }
		}
	    }

	  for( int c1 = 0 ; c1 < 3 ; ++c1 )
	    {
	      for( int c2 = 0 ; c2 < 3 ; ++c2)
		{
		  dest[ iQ(N,spin_i,spin_j,c1,c2,0,idx) ] -= res[c1][c2].real();
		  dest[ iQ(N,spin_i,spin_j,c1,c2,1,idx) ] -= res[c1][c2].imag();
		}
	    }

	  
	} // spin_j
    } // spin_i
}






void evaluate_special_prop_deriv_float( int threads, int N , int start, std::vector<void*> args , bool do_siteperm , int func_num )
{
  const int default_blocksize = 128;

  jumper_jit_stats_special(func_num);
  
  if (cuda_special_get_maxgridx() == -1)
    {
      std::cerr << "evaluate_dslash, cuda_special_maxgridx not set\n";
      exit(1);
    }
  
  int threads_per_block = default_blocksize;
  if ( cuda_special_get_blocksize().count( func_num ) > 0 )
    threads_per_block = cuda_special_get_blocksize()[ func_num ];
  
  std::pair<int,int> size = getBlockDim( N , threads_per_block );
  dim3 grid(  size.first , size.second , 1 );
  dim3 block( threads_per_block , 1 , 1 );

  // std::cout << "launching : grid( " << size.first << " , " << size.second << " , 1)   ";
  // std::cout << "launching : block( " << threads_per_block << " , 1 , 1 )\n";

  // 0 dest 
  // 1 u
  // 2,3 shift (off,recv)
  // 4 Q

  // 5,6 shift
  // 7 u
  // 8 Q

  float* dest = (float*)*(void**)args[0];

  int* goff[2];
  goff[0] = (int*)*(void**)args[2];
  goff[1] = (int*)*(void**)args[5];

  float* recv[2];
  recv[0] = (float*)*(void**)args[3];
  recv[1] = (float*)*(void**)args[6];

  float* Q1 = (float*)*(void**)args[4];
  float* Q2 = (float*)*(void**)args[8];

  float* u1 = (float*)*(void**)args[1];
  float* u2 = (float*)*(void**)args[7];

  int* site_perm = do_siteperm ? (int*)*(void**)args[9] : NULL;


  cuda_prop_deriv<float><<< grid , block >>>( threads, N , start ,
					      dest ,
					      Q1 , Q2,
					      u1 , u1,
					      goff[0] , recv[0] , 
					      goff[1] , recv[1] , 
					      do_siteperm, site_perm
					      );

}


void evaluate_special_prop_deriv_double( int threads, int N , int start, std::vector<void*> args , bool do_siteperm , int func_num )
{
  const int default_blocksize = 128;

  jumper_jit_stats_special(func_num);
  
  if (cuda_special_get_maxgridx() == -1)
    {
      std::cerr << "evaluate_dslash, cuda_special_maxgridx not set\n";
      exit(1);
    }
  
  int threads_per_block = default_blocksize;
  if ( cuda_special_get_blocksize().count( func_num ) > 0 )
    threads_per_block = cuda_special_get_blocksize()[ func_num ];
  
  std::pair<int,int> size = getBlockDim( N , threads_per_block );
  dim3 grid(  size.first , size.second , 1 );
  dim3 block( threads_per_block , 1 , 1 );

  // std::cout << "launching : grid( " << size.first << " , " << size.second << " , 1)   ";
  // std::cout << "launching : block( " << threads_per_block << " , 1 , 1 )\n";

  // 0 dest 
  // 1 u
  // 2,3 shift (off,recv)
  // 4 Q

  // 5,6 shift
  // 7 u
  // 8 Q

  double* dest = (double*)*(void**)args[0];

  int* goff[2];
  goff[0] = (int*)*(void**)args[2];
  goff[1] = (int*)*(void**)args[5];

  double* recv[2];
  recv[0] = (double*)*(void**)args[3];
  recv[1] = (double*)*(void**)args[6];

  double* Q1 = (double*)*(void**)args[4];
  double* Q2 = (double*)*(void**)args[8];

  double* u1 = (double*)*(void**)args[1];
  double* u2 = (double*)*(void**)args[7];

  int* site_perm = do_siteperm ? (int*)*(void**)args[9] : NULL;


  cuda_prop_deriv<double><<< grid , block >>>( threads, N , start ,
					      dest ,
					      Q1 , Q2,
					      u1 , u1,
					      goff[0] , recv[0] , 
					      goff[1] , recv[1] , 
					      do_siteperm, site_perm
					      );

}


#if 0
void linkage_hack()
{
  std::vector<void*> dummy;
  evaluate_special_prop_deriv<float> ( 0, 0 , 0, dummy , false , 0 );
  evaluate_special_prop_deriv<double>( 0, 0 , 0, dummy , false , 0 );
}
#endif


